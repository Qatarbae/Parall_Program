#include "hip/hip_runtime.h"
/* 10. Вычисление обратной матрицы */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

__device__ int max_element_index;

void print(double* matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%lf \t", matrix[i * n + j]);
        }
        printf("\n");
    }
}


__global__ void swap_lines(double* matrix, double* identity, int n, int row) {
    if (row == max_element_index) {
        return;
    }

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;

    double temp;
    for (; idx < n; idx += offset) {
	//__syncthreads(); 
        temp = matrix[row * n + idx];
        matrix[row * n + idx] = matrix[max_element_index * n + idx];
        matrix[max_element_index * n + idx] = temp;

        temp = identity[row * n + idx];
        identity[row * n + idx] = identity[max_element_index * n + idx];
        identity[max_element_index * n + idx] = temp;
    }
}

__global__ void max_in_column(double* matrix, int n, int x) {
    int max_index = x;
    double max_value = fabs(matrix[x * n + x]);
    double current_value;

    for (int i = x + 1; i < n; i++) {
	//__syncthreads(); 
        current_value = fabs(matrix[i * n + x]);
        if (current_value > max_value) {
            max_index = i;
            max_value = current_value;
        }
    }

    max_element_index = max_index;
}

__global__ void devide_identity(double* matrix, double* identity, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = gridDim.x * blockDim.x;
    int offsety = gridDim.y * blockDim.y;

    for (int i = idx; i < n; i += offsetx) {
        for (int j = idy; j < n; j += offsety) {
	   // __syncthreads();
            identity[i * n + j] /= matrix[i * n + i];
        }
    }
}

__global__ void devide_matrix(double* matrix, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetx = gridDim.x * blockDim.x;
    for (int i = idx; i < n; i += offsetx) {
        matrix[i * n + i] = 1.0;
    }
}

__global__ void subtract_below(double* matrix, double* identity, int n, int x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = gridDim.x * blockDim.x;
    int offsety = gridDim.y * blockDim.y;

    int i, j;
    double coeff;
    for (i = x + 1 + idx; i < n; i += offsetx) {
        coeff = matrix[i * n + x] / matrix[x * n + x];
        for (j = x + 1 + idy; j < n; j += offsety) {
		//__syncthreads();
            matrix[i * n + j] -= coeff * matrix[x * n + j];
        }
        for (j = idy; j < n; j += offsety) {
		//__syncthreads();
            identity[i * n + j] -= coeff * identity[x * n + j];
        }
    }
}

__global__ void nullify_below(double* matrix, int n, int x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetx = gridDim.x * blockDim.x;
    for (int i = x + 1 + idx; i < n; i += offsetx) {
        matrix[i * n + x] = 0.0;
    }
}

__global__ void subtract_above(double* matrix, double* identity, int n, int x) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offsetx = gridDim.x * blockDim.x;
    int offsety = gridDim.y * blockDim.y;

    int i, j;
    double coeff;
    for (i = x - 1 - idx; i >= 0; i -= offsetx) {
        coeff = matrix[i * n + x] / matrix[x * n + x];
        for (j = x - 1 - idy; j >= 0; j -= offsety) {
		//__syncthreads();
            matrix[i * n + j] -= coeff * matrix[x * n + j];
        }
        for (j = idy; j < n; j += offsety) {
		//__syncthreads();
            identity[i * n + j] -= coeff * identity[x * n + j];
        }
    }
}

__global__ void nullify_above(double* matrix, int n, int x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetx = gridDim.x * blockDim.x;
    for (int i = x - idx - 1; i >= 0; i -= offsetx) {
        matrix[i * n + x] = 0.0;
    }
}

void inverse_gpu(double* matrix, double* identity, int n) {
    dim3 BLOCKS_1D(16);
    dim3 THREADS_1D(32);
    dim3 BLOCKS_2D(16, 16);
    dim3 THREADS_2D(32, 32);

    double* dev_matrix;
    double* dev_identity;
    hipMalloc(&dev_matrix, sizeof(double) * n * n);
    hipMalloc(&dev_identity, sizeof(double) * n * n);
    hipMemcpy(dev_matrix, matrix, sizeof(double) * n * n,
        hipMemcpyHostToDevice);
    hipMemcpy(dev_identity, identity, sizeof(double) * n * n,
        hipMemcpyHostToDevice);



    for (int i = 0; i < n; i++) {
        max_in_column << <1, 1 >> > (dev_matrix, n, i);
        swap_lines << <BLOCKS_1D, THREADS_1D >> > (dev_matrix, dev_identity, n, i);
        subtract_below << <BLOCKS_2D, THREADS_2D >> > (dev_matrix, dev_identity, n, i);
        nullify_below << <BLOCKS_1D, THREADS_1D >> > (dev_matrix, n, i);
    }

    for (int i = n - 1; i >= 0; i--) {
        subtract_above << <BLOCKS_2D, THREADS_2D >> > (dev_matrix, dev_identity, n, i);
        nullify_above << <BLOCKS_1D, THREADS_1D >> > (dev_matrix, n, i);
    }

    devide_identity << <BLOCKS_2D, THREADS_2D >> > (dev_matrix, dev_identity, n);
    devide_matrix << <BLOCKS_1D, THREADS_1D >> > (dev_matrix, n);


    hipMemcpy(matrix, dev_matrix, sizeof(double) * n * n,
        hipMemcpyDeviceToHost);
    hipMemcpy(identity, dev_identity, sizeof(double) * n * n,
        hipMemcpyDeviceToHost);
    hipFree(dev_matrix);
    hipFree(dev_identity);
}


void swap_lines(double* matrix, double* identity, int n, int i, int j) {
    double temp;
    for (int k = 0; k < n; k++) {
        temp = matrix[i * n + k];
        matrix[i * n + k] = matrix[j * n + k];
        matrix[j * n + k] = temp;

        temp = identity[i * n + k];
        identity[i * n + k] = identity[j * n + k];
        identity[j * n + k] = temp;
    }
}

void pivotize(double* matrix, double* identity, int n, int row) {
    int max_index = row;
    double max_value = fabs(matrix[row * n + row]);
    double current_value;
    for (int i = row + 1; i < n; i++) {
        current_value = fabs(matrix[i * n + row]);
        if (current_value > max_value) {
            max_index = i;
            max_value = current_value;
        }
    }

    if (row != max_index) {
        swap_lines(matrix, identity, n, row, max_index);
    }
}

void devide(double* matrix, int n, int i, double denominator) {
    for (int j = 0; j < n; j++) {
        matrix[i * n + j] /= denominator;
    }
}

void subtract_below1(double* matrix, double* identity, int n, int x) {
    double coeff;
    for (int i = x + 1; i < n; i++) {
        coeff = matrix[i * n + x] / matrix[x * n + x];
        for (int j = x; j < n; j++) {
            matrix[i * n + j] -= coeff * matrix[x * n + j];
        }
        for (int j = 0; j < n; j++) {
            identity[i * n + j] -= coeff * identity[x * n + j];
        }
    }
}

void subtract_above1(double* matrix, double* identity, int n, int x) {
    double coeff;
    for (int i = x - 1; i >= 0; i--) {
        coeff = matrix[i * n + x] / matrix[x * n + x];
        for (int j = x; j >= 0; j--) {
            matrix[i * n + j] -= coeff * matrix[x * n + j];
        }
        for (int j = 0; j < n; j++) {
            identity[i * n + j] -= coeff * identity[x * n + j];
        }
    }
}

void inverse1(double* matrix, double* identity, int n) {
    for (int i = 0; i < n - 1; i++) {
        pivotize(matrix, identity, n, i);

        subtract_below1(matrix, identity, n, i);
    }

    for (int i = n - 1; i > 0; i--) {
        subtract_above1(matrix, identity, n, i);
    }

    for (int i = 0; i < n; i++) {
        devide(identity, n, i, matrix[i * n + i]);
        devide(matrix, n, i, matrix[i * n + i]);
    }
}

int** CreateArray(int N)
{
    int i, j;
    int** arr = (int**)malloc(N * sizeof(int*));
    for (i = 0; i < N; i++)
        arr[i] = (int*)malloc(N * sizeof(int));

    for (i = 0; i < N; i++)
        for (j = 0; j < N; j++)
            arr[i][j] = 0;

    return arr;
}

int main() {
    int n;
    int** arr = 0;
	char text[30];
	printf("input file: ");
	scanf("%s", &text);
    FILE* f = fopen(text, "rb");
    if (f) {
        printf("\nПроизводится считывание матрицы из файла. Пожалуйста, подождите...\n");
        // Находим размерность матрицы
        int str, str2;
        fread(&str, sizeof(int), 1, f);
        fread(&str2, sizeof(int), 1, f);
        n = str;
	printf("%d\n", n);
        arr = CreateArray(str); // Создание матрицы по считанным размерам
                              // Запись в матрицу значений из файла
        for (int i = 0; i < str; i++) {
            for (int j = 0; j < str2; j++)
            {
                fread(&arr[i][j], sizeof(int), 1, f);
            }
        }
       /* for (int i = 0; i < str; i++) {
            for (int j = 0; j < str2; j++)
            {
                printf("%d ",arr[i][j]);
            }
            printf("\n");
        }*/
        fclose(f);
    }
    else if (!f) {
        printf("File error.");
        return 1;
    }

    double* matrix = (double*)malloc(n * n * sizeof(double));
    double* identity = (double*)malloc(n * n * sizeof(double));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            matrix[i * n + j] = (double)arr[i][j];
        }
    }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            identity[i * n + j] = i == j ? 1.0 : 0.0;
        }
    }

   //Структуры для сохранения определенного времени
   struct timespec mt1, mt2; 
   
   //Определяем текущее время
   clock_gettime (CLOCK_REALTIME, &mt1);

    inverse_gpu(matrix, identity, n);

    clock_gettime (CLOCK_REALTIME, &mt2);
    double t = (mt2.tv_sec-mt1.tv_sec)*1000+(mt2.tv_nsec-mt1.tv_nsec)/1000000;
    printf ("Parall mls: %lf\n", t);
	printf("Posled \n");
    double* matrix1 = (double*)malloc(n * n * sizeof(double));
    double* identity1 = (double*)malloc(n * n * sizeof(double));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            matrix1[i * n + j] = (double)arr[i][j];
        }
    }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            identity1[i * n + j] = i == j ? 1.0 : 0.0;
        }
    }
clock_gettime(CLOCK_REALTIME, &mt1);
	inverse1(matrix1, identity1, n);
	//print(identity1, n);
clock_gettime (CLOCK_REALTIME, &mt2);
 t  = (mt2.tv_sec-mt1.tv_sec)*1000+(mt2.tv_nsec-mt1.tv_nsec)/1000000;
    printf ("Parall mls: %lf\n", t);
double sum = 0;
    for (int i = 0; i < n; i++) {
	    for (int j = 0; j < n; j++) {
            sum+= abs(identity[i*n+j] - identity1[i*n+j]);
        }
    }
	printf("dif = %.16lf\n", sum);

 	FILE* out = fopen("Results_parall.txt", "w");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
	//identity[i*n+j] = round(identity[i*n+j]*1000)/1000;
	//fwrite(&identity[i*n+j], sizeof(double), 1, out);
	fprintf(out, "%lf ",identity[i*n+j]); 
}
fputc('\n', out);
}

FILE* out1 = fopen("Results_posled.txt", "w");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
	//identity1[i*n+j] = round(identity1[i*n+j]*1000)/1000;
        //fwrite(&identity1[i*n+j], sizeof(double), 1, out1);
	fprintf(out1, "%lf ",identity1[i*n+j]);
}
fputc('\n', out1);
}

	fclose(out);
	fclose(out1);
    free(matrix);
    free(identity);
    free(matrix1);
    free(identity1);

    return 0;
}

